// This program gets the SM ID for each CTA launched in a kernel
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// Simple kernel that just gets the SM Id for each TB
__global__ void get_smid(unsigned *sm_ids){
    // Calculate global threadID
    int block = blockIdx.x;
    
    // SM Id
    unsigned sm;

    // Only one thread needs to do the write
    if(threadIdx.x == 0){
        // Use PTX instruction to get the SM Id
        asm volatile("mov.u32 %0, %smid;" : "=r"(sm));

        // Write the block back to main memory
        sm_ids[block] = sm;
    
    }
}

int main(){
    // Number of TBs to launch (16 for this example)
    int TB = 1 << 4;

    // Threads per TB
    int threads = 1024;

    // Allocate space for the SM results
    unsigned *h_sm_ids = new unsigned[TB];
    unsigned *d_sm_ids;
    hipMalloc(&d_sm_ids, TB * sizeof(unsigned));

    // Call the kernel
    get_smid<<<TB, threads>>>(d_sm_ids);

    // Copy back the Ids
    hipMemcpy(h_sm_ids, d_sm_ids, TB * sizeof(unsigned), hipMemcpyDeviceToHost);

    // Print the SM mapping
    for(int i = 0; i < TB; i++){
        cout << "TB " << i << " mapped to SM " << h_sm_ids[i] << endl;
    }

    return 0;
}

