// This program shows the affect of vector memory accesses on
// throughput
// By: Nick from CoffeeBeforeArch

#include <hip/hip_runtime.h>
#include <algorithm>

using namespace std;

// Max blocks is 80 SMs with 16 TBs of size 128
// Fills every thread of ever SM in a TITAN V
#define MAX_BLOCKS 1280
#define THREADS 128

// Each thread copies N elements of an array
// Takes:
//  in:     Input array
//  out:    Output array
//  N:      Number of elements to copy
// Returns:
//  NA
__global__ void scalar_copy(int *in, int *out, int N){
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy N elements from in to out
    // Moves over length of the grid each iteration
    for(int i = tid; i < N; i += blockDim.x * gridDim.x){
        out[i] = in[i];
    }
}

// Each thread copies N elements of an array using vector 2
// Takes:
//  in:     Input array
//  out:    Output array
//  N:      Number of elements to copy
// Returns:
//  NA
__global__ void vector_2_copy(int *in, int *out, int N){
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy N elements from in to out
    // Moves over length of the grid each iteration
    for(int i = tid; i < N / 2; i += blockDim.x * gridDim.x){
        reinterpret_cast<int2*>(out)[i] = reinterpret_cast<int2*>(in)[i];
    }
}

// Each thread copies N elements of an array using vector 4
// Takes:
//  in:     Input array
//  out:    Output array
//  N:      Number of elements to copy
// Returns:
//  NA
__global__ void vector_4_copy(int *in, int *out, int N){
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy N elements from in to out
    // Moves over length of the grid each iteration
    for(int i = tid; i < N / 4; i += blockDim.x * gridDim.x){
        reinterpret_cast<int4*>(out)[i] = reinterpret_cast<int4*>(in)[i];
    }
}

// Launch scalar copy
// Takes:
//  in:     Pointer to input vector
//  out:    Pointer to output vector
//  N:      Number of elements
void launch_scalar(int *in, int *out, int N){
    // Number of blocks to launch (power of 2 so no padding)
    int BLOCKS = min(N / THREADS, MAX_BLOCKS);
    
    // Launch the kernel
    scalar_copy<<<BLOCKS, THREADS>>>(in, out, N);
}

// Launch vector 2 copy
// Takes:
//  in:     Pointer to input vector
//  out:    Pointer to output vector
//  N:      Number of elements
void launch_vector_2(int *in, int *out, int N){
    // Number of blocks to launch (power of 2 so no padding)
    int BLOCKS = min(N / THREADS / 2, MAX_BLOCKS);
    
    // Launch the kernel
    vector_2_copy<<<BLOCKS, THREADS>>>(in, out, N);
}

// Launch vector 4 copy
// Takes:
//  in:     Pointer to input vector
//  out:    Pointer to output vector
//  N:      Number of elements
void launch_vector_4(int *in, int *out, int N){
    // Number of blocks to launch (power of 2 so no padding)
    int BLOCKS = min(N / THREADS / 4, MAX_BLOCKS);
    
    // Launch the kernel
    vector_4_copy<<<BLOCKS, THREADS>>>(in, out, N);
}


int main(){
    // Starting number of elements is 4k
    int start = 1 << 12;
    // Ending size is 1G
    int end = 1 << 30;
    // Number of elements for each thread to load
    // Size of the array in bytes
    size_t bytes;

    // Host and device pointers
    int *h_in, *h_out;
    int *d_in, *d_out;

    // Test from 2^12 -> 2^29
    for(int i = start; i <= end; i *= 2){
        // Allocate host memory
        h_in = new int[i];
        h_out = new int[i];

        // Calculate bytes to allocate on GPU
        bytes = i * sizeof(int);

        // Allocate device memory
        hipMalloc(&d_in, bytes);
        hipMalloc(&d_out, bytes);

        // Copy data to GPU
        hipMemcpy(d_in, h_in, bytes, hipMemcpyHostToDevice);

        launch_scalar(d_in, d_out, i);
        //launch_vector_2(d_in, d_out, i);
        //launch_vector_4(d_in, d_out, i);

        // Copy data from GPU
        hipMemcpy(h_out, d_out, bytes, hipMemcpyDeviceToHost);

        // Free memory after use for re-allocation
        delete [] h_in;
        delete [] h_out;
        hipFree(d_in);
        hipFree(d_out);
    }

    return 0;
}
