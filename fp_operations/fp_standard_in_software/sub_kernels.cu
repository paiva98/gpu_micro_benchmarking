
#include <hip/hip_runtime.h>
// This file contains multiple arithmetic kernels to better understand
// how they translate into the machine instruction set
// By: Nick from CoffeeBeforeArch

// Subtraction kernels
// Char
__global__ void char_sub(char *a, char *b, char *c){
    *c = *a - *b;
}
// Integer
__global__ void int_sub(int *a, int *b, int *c){
    *c = *a - *b;
}
// Float
__global__ void float_sub(float *a, float *b, float *c){
    *c = *a - *b;
}
// Double
__global__ void double_sub(double *a, double *b, double *c){
    *c = *a - *b;
}
