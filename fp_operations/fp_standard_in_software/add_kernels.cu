
#include <hip/hip_runtime.h>
// This file contains multiple arithmetic kernels to better understand
// how they translate into the machine instruction set
// By: Nick from CoffeeBeforeArch

// Addition kernels
// Char
__global__ void char_add(char *a, char *b, char *c){
    *c = *a + *b;
}
// Integer
__global__ void int_add(int *a, int *b, int *c){
    *c = *a + *b;
}
// Float
__global__ void float_add(float *a, float *b, float *c){
    *c = *a + *b;
}
// Double
__global__ void double_add(double *a, double *b, double *c){
    *c = *a + *b;
}
