
#include <hip/hip_runtime.h>
// This file contains multiple arithmetic kernels to better understand
// how they translate into the machine instruction set
// By: Nick from CoffeeBeforeArch

// Multiplication kernels
// Char
__global__ void char_mul(char *a, char *b, char *c){
    *c = *a * *b;
}
// Integer
__global__ void int_mul(int *a, int *b, int *c){
    *c = *a * *b;
}
// Float
__global__ void float_mul(float *a, float *b, float *c){
    *c = *a * *b;
}
// Double
__global__ void double_mul(double *a, double *b, double *c){
    *c = *a * *b;
}
