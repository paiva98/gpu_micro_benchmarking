
#include <hip/hip_runtime.h>
// This file contains multiple arithmetic kernels to better understand
// how they translate into the machine instruction set
// By: Nick from CoffeeBeforeArch

// Division kernels
// Char
__global__ void char_div(char *a, char *b, char *c){
    *c = *a / *b;
}
// Integer
__global__ void int_div(int *a, int *b, int *c){
    *c = *a / *b;
}
// Float
__global__ void float_div(float *a, float *b, float *c){
    *c = *a / *b;
}
// Double
__global__ void double_div(double *a, double *b, double *c){
    *c = *a / *b;
}
