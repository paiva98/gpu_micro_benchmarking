// This program show's off how CUDA memory copies go through the
// L2 Cache
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <stdlib.h>

// Simple vector addition kernel with easily calculable number of reads
__global__ void vectorAdd(int *a, int *b, int *c, int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < N){
        c[tid] = a[tid] | b[tid];
    }
}

int main(){
    // Number of elements 
    int N = 1 << 10;
    size_t bytes = N * sizeof(int);

    // Host arrays
    int *h_a = new int[N];
    int *h_b = new int[N];
    int *h_c = new int[N];

    for(int i = 0; i < N; i++){
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }

    // Device Arrays
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy data over
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // TB Size and Grid Size
    int THREADS = 256;
    int GRID = (N + THREADS - 1) / THREADS;

    vectorAdd<<<GRID, THREADS>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    
    return 0;
}
